#include "hip/hip_runtime.h"
/*! \file mcea_async.cu
  Main algorithm. Does all the memory management and starts the kernels.
*/
#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <string>
#include <vector>

// own header files
#include "../util/output.cuh"
#include "../util/neighbor.cuh"
#include "../util/random.cuh"
#include "../util/dtlz.h"
#include "../util/weighting.cuh"
#include "../util/error.h"
#include "config.h"

/*! \brief McEA kernel

  This kernel runs the whole algorithm. All data structures have to be set up prior to this.
  It uses the population and performs GENERATIONS generations, consisting of pairing, crossover, mutation, evaluation, and selection on it.
  At the end the population contains the optimized individuals.

  \param[in,out] population an array containing all parameters of the whole population.
  \param[out] objectives an array containing all objective values 
  \param[in] rng_state the initialized state of the PRNG to use
*/
__global__ void mcea( float *population, float *objectives, hiprandStatePhilox4_32_10_t *rng_state ) {
  __shared__ float offspring[PARAMS * BLOCKSIZE];
  __shared__ float offspring_fit[OBJS * BLOCKSIZE];
  __shared__ float weights[OBJS * BLOCKSIZE];
  hiprandStatePhilox4_32_10_t rng_local;
  float4_union randn_neigh_1, randn_neigh_2, randn_xover_point;
  int4_union randn_mut_count;
  double fit_parent;

  // global indices
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = x + y * (POP_WIDTH + 1);
  // blockwise indices
  int block_idx = (blockDim.x * threadIdx.y + threadIdx.x);

  // init RNG, fitness and weights
  if( x < POP_WIDTH + 1 && y < POP_WIDTH ) {
    rng_local = *(rng_state + idx);
    // ### evaluation ###
    dtlz( population+idx, objectives+idx, PARAMS, OBJS, POP_SIZE );
    calc_weights(x, y, weights + block_idx, BLOCKSIZE);
    fit_parent =  weighted_fitness( objectives + idx, weights + block_idx, POP_SIZE );
  }

  // main loop
#if STOPTYPE == GENERATIONS
  // stop after number of generations
  for (size_t g = 0; g < STOPVALUE; g++) {
#elif STOPTYPE == TIME
    if(idx == 0)
      printf( "STOPTYPE: TIME is not possible for async computation. doing just one generation.\n" );
    {
      int g = 0;
#else
    if(idx == 0)
      printf( "no valid STOPTYPE. doing just one generation.\n" );
    {
      int g = 0;
#endif

    if( x < POP_WIDTH + 1 && y < POP_WIDTH ) {

      // ### generate random numbers ###
      // pre-generate for 4 generations
      if(g % 4 == 0) {
        randn_neigh_1.vec = hiprand_uniform4( &rng_local );
        randn_neigh_2.vec = hiprand_uniform4( &rng_local );
        randn_xover_point.vec = hiprand_uniform4( &rng_local );
        randn_mut_count.vec = hiprand_poisson4( &rng_local, LAMBDA );

      }

      // ### pairing ###
      // random neighbors
      int neighbor_1 = get_neighbor( x, y, trans_uniform_int( randn_neigh_1.arr[g%4], N_WIDTH * N_WIDTH ) );
      int neighbor_2 = get_neighbor( x, y, trans_uniform_int( randn_neigh_2.arr[g%4], N_WIDTH * N_WIDTH ) );

      // compare neighbors
      double fit_1 =  weighted_fitness( objectives + neighbor_1, weights + block_idx, POP_SIZE );
      double fit_2 =  weighted_fitness( objectives + neighbor_2, weights + block_idx, POP_SIZE );
      int neighbor_sel = (fit_1 < fit_2)? neighbor_1 : neighbor_2;

      if( idx == 0 && VERBOSE )
        printf("x: %d, y: %d, n1: %3d(%.3f), n2: %3d(%.3f), sel: %3d\n", x, y, neighbor_1, fit_1, neighbor_2, fit_2, neighbor_sel);

      if( idx == 0 && VERBOSE ) {
        printf( "original: " );
        for (size_t i = 0; i < PARAMS; i++)
          printf( "%.2f, ", population[idx + i * POP_SIZE] );
        printf( "\n" );
        for (size_t i = 0; i < OBJS; i++)
          printf( "%.2f, ", objectives[i + idx * POP_SIZE] );
        printf( "\n" );
      }

      // ### crossover ###
      // == one-point crossover
      int x_over_point = trans_uniform_int( randn_xover_point.arr[g%4], PARAMS );
      if( idx == 0 && VERBOSE )
        printf( "xover: %d\n", x_over_point );

      for (size_t i = 0; i < PARAMS; i++)
        offspring[block_idx + BLOCKSIZE * i] = (i<x_over_point) ? population[idx + i * POP_SIZE] : population[neighbor_sel + i * POP_SIZE];

      if( idx == 0 && VERBOSE ) {
        printf( "crossover: " );
        for (size_t i = 0; i < PARAMS; i++)
          printf( "%.2f, ", offspring[block_idx + BLOCKSIZE * i] );
        printf( "\n" );
      }
      // ### mutation ###
      // == uniform mutation
      int num_mutations = randn_mut_count.arr[g%4];
      if( idx == 0 && VERBOSE )
        printf( "mut: %d\n", num_mutations );

      for (size_t i = 0; i < num_mutations; i++) {
        int mut_location = trans_uniform_int( hiprand_uniform(&rng_local), PARAMS );
        offspring[block_idx + BLOCKSIZE * mut_location] = hiprand_uniform( &rng_local );
      }

      if( idx == 0 && VERBOSE ) {
        printf( "mutated: " );
        for (size_t i = 0; i < PARAMS; i++)
          printf( "%.2f, ", offspring[block_idx + BLOCKSIZE * i] );
        printf( "\n" );
      }

      // ### selection ###
      // == select if better

      // evaluate the offspring
      dtlz( offspring + block_idx , offspring_fit + block_idx, PARAMS, OBJS, BLOCKSIZE );

      if( idx == 0 && VERBOSE ) {
        printf( "offspring fit: " );
        for (size_t i = 0; i < OBJS; i++)
          printf( "%.2f, ", offspring_fit[block_idx + BLOCKSIZE * i] );
        printf( "\n" );
      }

      // compare and copy
      fit_2 =  weighted_fitness( offspring_fit + block_idx, weights + block_idx, BLOCKSIZE );

      if( idx == 0 && VERBOSE ) {
        printf( "offspring weight: %.5lf\n", fit_2 );
      }

      if(fit_2 < fit_parent) {
        for (size_t i = 0; i < PARAMS; i++)
          population[idx + i * POP_SIZE] = offspring[block_idx + BLOCKSIZE * i];
        for (size_t i = 0; i < OBJS; i++)
          objectives[idx + i * POP_SIZE] = offspring_fit[block_idx + BLOCKSIZE * i];
        fit_parent = fit_2;
      }

      if( idx == 0 && VERBOSE ) {
        printf( "new ind: " );
        for (size_t i = 0; i < PARAMS; i++)
          printf( "%.2f, ", population[idx + i * POP_SIZE] );
        printf( "\n" );
        for (size_t i = 0; i < OBJS; i++)
          printf( "%.2f, ", objectives[idx + i * POP_SIZE] );
        printf( "\n" );
      }
    }

    // sync the block after every generation
    __syncthreads();
  }

  return;
}

/*! \brief main function

  Classic main function. It allocates all memory, generates the population, starts the kernel and collects the results.
  All parameter changes are made via the \#define statements
*/
int main(int argc, char *argv[]) {

  // get the output folder the run number and type
  string folder = "";
  string run = "0";
  if(argc > 1) {
    folder = argv[1];
    run = argv[2];
  }

  run = string("async_") + run;

  // allocate memory
  float *population_h = (float *)malloc( POP_SIZE * PARAMS * sizeof(float) );
  float *objectives_h = (float *)malloc( POP_SIZE * OBJS * sizeof(float) );
  float *population_d;
  float *objectives_d;

  hiprandStatePhilox4_32_10_t *d_state;
  ERR( hipMalloc( &d_state, POP_SIZE * sizeof(hiprandStatePhilox4_32_10_t) ) );
  ERR( hipMalloc( (void**)&population_d, POP_SIZE * PARAMS * sizeof(float) ) );
  ERR( hipMalloc( (void**)&objectives_d, POP_SIZE * OBJS * sizeof(float) ) );

  // setup random generator
  unsigned long seed = clock();
  rand_init<<<POP_SIZE / 1024 + 1, 1024>>>( d_state, seed );

  // create random population
  srand( time( NULL ) );
  for (size_t i = 0; i < POP_SIZE; i++) {
    for (size_t j = 0; j < PARAMS; j++) {
      population_h[i * PARAMS + j] = randomFloat();
    }
  }

  // copy data to GPU
  ERR( hipMemcpy( population_d, population_h, POP_SIZE * PARAMS * sizeof(float), hipMemcpyHostToDevice ) );

  // capture the start time
  hipEvent_t     start, stop;
  ERR( hipEventCreate( &start ) );
  ERR( hipEventCreate( &stop ) );
  ERR( hipEventRecord( start, 0 ) );

  // start the kernel
  dim3 dimBlock(BLOCKDIM, BLOCKDIM);
  dim3 dimGrid(ceil((POP_WIDTH + 1) / (float)BLOCKDIM) , ceil(POP_WIDTH / (float)BLOCKDIM));
  mcea<<<dimGrid, dimBlock>>>( population_d, objectives_d, d_state );

  // get stop time, and display the timing results
  ERR( hipEventRecord( stop, 0 ) );
  ERR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  ERR( hipEventElapsedTime( &elapsedTime, start, stop ) );
  printf( "duration:  %f ms\n", elapsedTime );

  // copy data from GPU
  ERR( hipMemcpy( population_h, population_d, POP_SIZE * PARAMS * sizeof(float), hipMemcpyDeviceToHost ) );
  ERR( hipMemcpy( objectives_h, objectives_d, POP_SIZE * OBJS * sizeof(float), hipMemcpyDeviceToHost ) );

  // write the results to file
  write_objectives( objectives_h, folder, run );
  write_info( elapsedTime, folder, run );

  // free resources
  free( population_h );
  free( objectives_h );
  ERR( hipEventDestroy( start ) );
  ERR( hipEventDestroy( stop ) );

  ERR( hipFree( population_d ) );
  ERR( hipFree( objectives_d ) );
  ERR( hipFree( d_state ) );
}
