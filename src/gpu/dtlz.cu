#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"
#include "config.h"
#include "error.h"

/*! \file dtlz.cu
  This module contains all used DTLZ functions. Definition of the functions can be found in:
  Deb, K., Thiele, L., Laumanns, M., & Zitzler, E. (2005). Scalable test problems for evolutionary multiobjective optimization (pp. 105-145). Springer London.
*/


#if DTLZ_NUM == 0
/*!
  Test function. Performs the sum of all params and multiplies it with the respective objectives params.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

    float param_sum = 0.0;
    for (size_t i = 0; i < param_size; i++) {
      param_sum += params[i * step_size];
    }

    for (size_t i = 0; i < obj_size; i++) {
      objectives[i * step_size] = param_sum * i;
    }

    return;
  }

#elif DTLZ_NUM == 1
/*! \brief Function of the DTLZ1 multicriterial optimization problem

  Calculates the objectives for the DTLZ1 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

		double g = 0.0;
		for (int i = obj_size - 1; i < param_size; i++) {
			g += powf(params[i * step_size] - 0.5, 2.0)
					- cosf(20.0 * HIP_PI_F * (params[i * step_size] - 0.5));
		}
		g = 0.5 * (1.0 + 100.0 * (param_size - obj_size + 1 + g));

    // first iteration is different
    double f = g;
		for (int j = 0; j < obj_size - 1; j++) {
			f *= params[j * step_size];
		}
    objectives[0] = f;

    // all others have 1 additional step
		for (int i = 1; i < obj_size; i++) {
			f = g;

			for (int j = 0; j < obj_size - i - 1; j++) {
				f *= params[j * step_size];
			}
			f *= 1 - params[(obj_size - i - 1) * step_size];

      objectives[i * step_size] = f;
		}

    return;
}

#elif DTLZ_NUM == 2
/*! \brief Function of the DTLZ2 multicriterial optimization problem

  Calculates the objectives for the DTLZ2 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

  double g = 0.0;
  for (int i = obj_size - 1; i < param_size; i++)
    g += powf(params[i * step_size] - 0.5, 2.0);

  // different first iteration
  double f = (1 + g);
  for (int j = 0; j < obj_size - 1; j++)
    f *= cosf(params[j * step_size] * HIP_PI_F / 2);
  objectives[0] = f;

  for (int i = 1; i < obj_size; i++) {
    f = (1 + g);
    for (int j = 0; j < obj_size - i - 1; j++)
      f *= cosf(params[j * step_size] * HIP_PI_F / 2);

    f *= sinf(params[(obj_size - i - 1) * step_size] * HIP_PI_F / 2);

    objectives[i * step_size] = f;
  }
}

#elif DTLZ_NUM == 3
/*! \brief Function of the DTLZ3 multicriterial optimization problem

  Calculates the objectives for the DTLZ3 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

	double g = 0.0;
	for (int i = obj_size - 1; i < param_size; i++) {
		g += powf(params[i * step_size] - 0.5, 2.0)
				- cosf(20.0 * HIP_PI_F * (params[i * step_size] - 0.5));
	}
	g = 1.0 + 100.0 * (param_size - obj_size + 1 + g);

  // different first iteration
  double f = g;
  for (int j = 0; j < obj_size - 1; j++)
    f *= cosf(params[j * step_size] * HIP_PI_F / 2);
  objectives[0] = f;

  for (int i = 1; i < obj_size; i++) {
    f = g;
    for (int j = 0; j < obj_size - i - 1; j++)
      f *= cosf(params[j * step_size] * HIP_PI_F / 2);

    f *= sinf(params[(obj_size - i - 1) * step_size] * HIP_PI_F / 2);

    objectives[i * step_size] = f;
  }
}

#elif DTLZ_NUM == 4
/*! \brief Function of the DTLZ4 multicriterial optimization problem

  Calculates the objectives for the DTLZ4 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

  double g = 0.0;
  double alpha = 100.0;
  for (int i = obj_size - 1; i < param_size; i++)
    g += powf(params[i * step_size] - 0.5,2);

  // different first iteration
  double f = (1 + g);
  for (int j = 0; j < obj_size - 1; j++)
    f *= cos( powf(params[j * step_size], alpha) * HIP_PI_F / 2);
  objectives[0] = f;

  for (int i = 1; i < obj_size; i++) {
    f = (1 + g);
    for (int j = 0; j < obj_size - i - 1; j++)
      f *= cos( powf(params[j * step_size], alpha) * HIP_PI_F / 2);

    f *= sin( powf(params[(obj_size - i - 1) * step_size], alpha) * HIP_PI_F / 2);

    objectives[i * step_size] = f;
  }
}

#elif DTLZ_NUM == 5
/*! \brief Function of the DTLZ5 multicriterial optimization problem

  Calculates the objectives for the DTLZ5 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

  float g = 0.0;
  float t = 0.0;
  float theta[OBJS-1];

  for (int i = obj_size - 1; i < param_size; i++) {
    g += powf(params[i * step_size]-0.5,2);
  }

  t = HIP_PI_F /(4 * (1 + g));

  theta[0]= (HIP_PI_F / 2) * params[0];
  for (int i = 1; i < obj_size - 1 ; i++)
    theta[i]=  t * (1 + 2 * g * params[i * step_size]);

  double f = 1 + g;
  for (int j = 0; j < obj_size - 1; j++)
      f *= cosf(theta[j]);
  objectives[0] = f;

  for (int i = 1; i < obj_size; i++) {
    f = (1 + g);
    for (int j = 0; j < obj_size - i - 1; j++)
      f *= cosf(theta[j]);

    f *= sinf(theta[obj_size - i - 1]);

    objectives[i * step_size] = f;
  }
}

#elif DTLZ_NUM == 6
/*! \brief Function of the DTLZ6 multicriterial optimization problem

  Calculates the objectives for the DTLZ6 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size ) {

  float g = 0.0;
  float t = 0.0;
  float theta[OBJS-1];

  for (int i = obj_size - 1; i < param_size; i++)
    g += powf(params[i * step_size],0.1);

  t = HIP_PI_F /(4 * (1 + g));

  theta[0]= (HIP_PI_F / 2) * params[0];
  for (int i = 1; i < obj_size - 1 ; i++)
    theta[i]=  t * (1 + 2 * g * params[i * step_size]);

  double f = 1 + g;
  for (int j = 0; j < obj_size - 1; j++)
      f *= cosf(theta[j]);
  objectives[0] = f;

  for (int i = 1; i < obj_size; i++) {
    f = (1 + g);
    for (int j = 0; j < obj_size - i - 1; j++)
      f *= cosf(theta[j]);

    f *= sinf(theta[obj_size - i - 1]);

    objectives[i * step_size] = f;
  }
}

#elif DTLZ_NUM == 7
/*! \brief Function of the DTLZ7 multicriterial optimization problem

  Calculates the objectives for the DTLZ7 problem [deb2005scalable], given an array of parameters.

  \param params pointer to array of param values
  \param objectives pointer to objective array
  \param param_size number of elements in the param array
  \param obj_size number of elements in the objective array
  \param step_size number of elements to skip between 2 parameters (should be the number of threads)
*/
__device__ void dtlz( float *params, float *objectives, int param_size, int obj_size, int step_size) {

        float g = 0.0;
        float h = 0.0;

        for (int i = obj_size - 1; i < param_size; i++) {
            g += params[i * step_size];
        }
        g= 2 + ( 9 * g ) / (param_size - obj_size + 1);


        for (int i = 0; i < obj_size - 1 ; i++)
            objectives[i * step_size] = params[i * step_size];

        for (int i = 0 ; i < obj_size - 1; i++)
            h += params[i * step_size] / g * (1 + sinf(3 * HIP_PI_F * params[i * step_size]));
        h = obj_size - h;

        objectives[(obj_size-1) * step_size] =  g * h;
}
#endif
