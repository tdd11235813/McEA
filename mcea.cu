#include "hip/hip_runtime.h"
/*! \file mcea.cu
  Main algorithm. Does all the memory management and starts the kernels.
*/
#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

// own header files
#include "error.h"
#include "util.h"
#include "dtlz.cuh"

#define POP_WIDTH 10
#define POP_SIZE (POP_WIDTH * (POP_WIDTH + 1))
#define PARAMS 100
#define OBJS 3
#define N_RAD 1
#define N_WIDTH (2 * N_RAD + 1)
#define P_MUT 0.01
#define LAMBDA (P_MUT * PARAMS)

/*! \brief neighbor calculation

  For a given neighbor index this calculates the neighbors global position realtive to the original individual.
  The neighbor index is a number representing the position of the neighbor in the neighborhood of the individual.
  It is organized rowwise starting at the top-left individual in the neighborhood (index = 0). When there is an
  overflow in any of the directions, the global index will be wrapped around to the other side of the population.

  \param x the x position of the original individual
  \param y the y position of the original individual
  \param neighbor_index the position of the neighbor relative to the original. Allowed values depend on the population size.
*/
__device__ int get_neighbor(int x, int y, int neighbor_index) {
  // 2D indices
  int n_x = (x + neighbor_index % N_WIDTH - N_RAD + POP_WIDTH + 1) % (POP_WIDTH + 1);
  int n_y = (y + neighbor_index / N_WIDTH - N_RAD + POP_WIDTH) % POP_WIDTH;

  // global index
  return n_x + n_y * (POP_WIDTH + 1);
}

/*! \brief init PRNG

Initializes the pseudo random number generator.

\param state the data structure for the RNG state
\param seed the seed with which to init the RNG
*/
__global__ void rand_init( hiprandState *state ) {
  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  unsigned long long seed = (unsigned long long) clock64();
  hiprand_init(seed, idx, 0, &state[idx]);
}

/* \brief generates a random uniform int

  Draws from a uniform distribution in [0, 1] and converts it to an integer in the range [0, values-1].

  \param state the PRNG state to use
  \param values the number of possible values for the uniform distribution
*/
__device__ int rnd_uniform_int( hiprandState *state, int values ) {

    return (int)truncf( hiprand_uniform( state ) * ( values - 0.000001) );
}

/* \brief calculates the weighted fitness

Takes the objective values of the individual at idx and calculates its fitness.
The specific weights for the individual at location x,y in the population are used for weighting.
! This only works for 3 objectives for now !
TODO: for real world problems use the weighted tchebychev method (use utopia vector)

\param objectives pointer to the objective values of the individual
\param x the x location of the weighting basis (does not have to be the same ind the objectives are from)
\param y the y location of the weighting basis (does not have to be the same ind the objectives are from)
*/
__device__ float weighted_fitness( float *objectives, int x, int y ) {
  // this decides if the individual is on the mirrored side of the population
  // and gives the correction factor for the weights
  int mirror = (x + y < POP_WIDTH)? false : true;

  float offset =  (mirror) ? 0.25 : 0.0;
  int _x  =  (mirror) ? POP_WIDTH - y - 1 : x;
  int _y  =  (mirror) ? POP_WIDTH - x : y;

  // calculate the fitness
  return \
      objectives[0] * (1 - (_x+offset)/(POP_WIDTH-0.5) - (_y+offset)/(POP_WIDTH-0.5)) \
    + objectives[1] * (_x+offset)/(POP_WIDTH-0.5) \
    + objectives[2] * (_y+offset)/(POP_WIDTH-0.5);

}

/*! \brief main kernel

  This kernel runs the whole algorithm. All data structures have to be set up for this.
  TODO: implement algorithm
  \param population an array containing all parameters of the whole population.
  \param objectives an array containing all objective values (there will be written some new ones)
  \param utopia_vec a vector containing the best values for each single objective
  \param rng_state the initialized state of the PRNG to use
*/
__global__ void mcea( float *population, float *objectives, float *utopia_vec, hiprandState *rng_state ) {
  float offspring[PARAMS];
  float offspring_fit[OBJS];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = x + y * (POP_WIDTH + 1);


  if( idx < POP_SIZE ) {
    // ### evaluation ###
    dtlz1( population+idx*PARAMS, objectives+idx*OBJS, PARAMS, OBJS );

    // ### pairing ###
    // random neighbors
    int neighbor_1 = get_neighbor( x, y, rnd_uniform_int( rng_state + idx, N_WIDTH * N_WIDTH ) );
    int neighbor_2 = get_neighbor( x, y, rnd_uniform_int( rng_state + idx, N_WIDTH * N_WIDTH ) );

    // compare neighbors
    float fit_1 = weighted_fitness( objectives + neighbor_1 * OBJS, x, y );
    float fit_2 = weighted_fitness( objectives + neighbor_2 * OBJS, x, y );
    int neighbor_sel = (fit_1 < fit_2)? neighbor_1 : neighbor_2;

    if( idx == 99 )
      printf("x: %d, y: %d, n1: %3d(%.3f), n2: %3d(%.3f), sel: %3d\n", x, y, neighbor_1, fit_1, neighbor_2, fit_2, neighbor_sel);

    if( idx == 99 ) {
      printf( "original: " );
      for (size_t i = 0; i < PARAMS; i++)
        printf( "%.2f, ", population[i + idx * PARAMS] );
      printf( "\n" );
      for (size_t i = 0; i < OBJS; i++)
        printf( "%.2f, ", objectives[i + idx * OBJS] );
      printf( "\n" );
    }
    // ### crossover ###
    // == one-point crossover
    int x_over_point = rnd_uniform_int( rng_state + idx, PARAMS );
    if( idx == 99 )
      printf( "xover: %d\n", x_over_point );

    for (size_t i = 0; i < PARAMS; i++)
      offspring[i] = (i<x_over_point) ? population[i + idx * PARAMS] : population[i + neighbor_sel * PARAMS];

    if( idx == 99 ) {
      printf( "crossover: " );
      for (size_t i = 0; i < PARAMS; i++)
        printf( "%.2f, ", offspring[i] );
      printf( "\n" );
    }
    // ### mutation ###
    // == uniform mutation
    int num_mutations = hiprand_poisson( rng_state + idx, LAMBDA );
    if( idx == 99 )
      printf( "mut: %d\n", num_mutations );

    for (size_t i = 0; i < num_mutations; i++) {
      int mut_location = rnd_uniform_int( rng_state + idx, PARAMS );
      offspring[mut_location] = hiprand_uniform( rng_state + idx );
    }

    if( idx == 99 ) {
      printf( "mutated: " );
      for (size_t i = 0; i < PARAMS; i++)
        printf( "%.2f, ", offspring[i] );
      printf( "\n" );
    }

    // ### selection ###
    // == select if better

    // evaluate the offspring
    dtlz1( offspring, offspring_fit, PARAMS, OBJS );

    if( idx == 99 ) {
      printf( "offspring fit: " );
      for (size_t i = 0; i < OBJS; i++)
        printf( "%.2f, ", offspring_fit[i] );
      printf( "\n" );
    }

    // compare and copy
    fit_1 = weighted_fitness( objectives + idx * OBJS, x, y );
    fit_2 = weighted_fitness( offspring_fit, x, y );

    if(fit_2 < fit_1) {
      for (size_t i = 0; i < PARAMS; i++)
        population[i + idx * PARAMS] = offspring[i];
      for (size_t i = 0; i < OBJS; i++)
        objectives[i + idx * OBJS] = offspring_fit[i];
    }

    if( idx == 99 ) {
      printf( "new ind: " );
      for (size_t i = 0; i < PARAMS; i++)
        printf( "%.2f, ", population[i + idx * PARAMS] );
      printf( "\n" );
      for (size_t i = 0; i < OBJS; i++)
        printf( "%.2f, ", objectives[i + idx * OBJS] );
      printf( "\n" );
    }

    __syncthreads();
  }

  return;
}

/*! \brief main function

  Classic main function. It allocates all memory, generates the population, starts the kernel and collects the results.
  All parameters changes are made via the #define statements
*/
int main() {
  // allocate memory
  float population_h[POP_SIZE][PARAMS];
  float objectives_h[POP_SIZE][OBJS];
  float utopia_vec_h[OBJS];
  float *population_d;
  float *objectives_d;
  float *utopia_vec_d;

  hiprandState *d_state;
  ERR( hipMalloc( &d_state, POP_SIZE * sizeof(hiprandState) ) );
  ERR( hipMalloc( (void**)&population_d, POP_SIZE * PARAMS * sizeof(float) ) );
  ERR( hipMalloc( (void**)&objectives_d, POP_SIZE * OBJS * sizeof(float) ) );
  ERR( hipMalloc( (void**)&utopia_vec_d, OBJS * sizeof(float) ) );

  // setup random generator
  rand_init<<<1, POP_SIZE>>>( d_state );

  // create random population
  srand( time( NULL ) );
  for (size_t i = 0; i < POP_SIZE; i++) {
    for (size_t j = 0; j < PARAMS; j++) {
      population_h[i][j] = randomFloat();
      //population_h[i][j] = ((float)i)/PARAMS;
    }
  }

  // copy data to GPU
  ERR( hipMemcpy( population_d, population_h, POP_SIZE * PARAMS * sizeof(float), hipMemcpyHostToDevice ) );

  // capture the start time
  hipEvent_t     start, stop;
  ERR( hipEventCreate( &start ) );
  ERR( hipEventCreate( &stop ) );
  ERR( hipEventRecord( start, 0 ) );

  // start the kernel
  dim3 dimBlock(POP_WIDTH + 1, POP_WIDTH);
  mcea<<<1, dimBlock>>>( population_d, objectives_d, utopia_vec_d, d_state );

  // get stop time, and display the timing results
  ERR( hipEventRecord( stop, 0 ) );
  ERR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  ERR( hipEventElapsedTime( &elapsedTime, start, stop ) );
  printf( "Time to generate:  %f ms\n", elapsedTime );

  // copy data from GPU
  ERR( hipMemcpy( population_h, population_d, POP_SIZE * PARAMS * sizeof(float), hipMemcpyDeviceToHost ) );
  ERR( hipMemcpy( objectives_h, objectives_d, POP_SIZE * OBJS * sizeof(float), hipMemcpyDeviceToHost ) );
  ERR( hipMemcpy( utopia_vec_h, utopia_vec_d, OBJS * sizeof(float), hipMemcpyDeviceToHost ) );

  ERR( hipEventDestroy( start ) );
  ERR( hipEventDestroy( stop ) );

  // print some solutions
  printVector( population_h[0], PARAMS );
  printVector( objectives_h[0], OBJS );

  // free resources
  ERR( hipFree( population_d ) );
  ERR( hipFree( objectives_d ) );
  ERR( hipFree( utopia_vec_d ) );
}
