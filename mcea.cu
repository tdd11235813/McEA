#include "hip/hip_runtime.h"
/*! \file mcea.cu
  Main algorithm. Does all the memory management and starts the kernels.
*/
#include "hip/hip_runtime.h"

// own header files
#include "error.h"
#include "util.h"
#include "dtlz.cuh"

#define POP_WIDTH 10
#define POP_SIZE ((POP_WIDTH * (POP_WIDTH + 1)) / 2)
#define PARAMS 50
#define OBJS 3

/*! \brief main kernel

  This kernel runs the whole algorithm. All data structures have to be set up for this.
  TODO: implement algorithm
  \param population an array containing all parameters of the whole population.
  \param objectives an array containing all objective values (there will be written some new ones)
  \param utopia_vec a vector containing the best values for each single objective
*/
__global__ void mcea( float *population, float *objectives, float *utopia_vec ) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if( idx < POP_SIZE ) {
    testObjSum( population+idx, objectives+idx, PARAMS, OBJS );
  }

  return;
}

/*! \brief main function

  Classic main function. It allocates all memory, generates the population, starts the kernel and collects the results.
  All parameters changes are made via the #define statements
*/
int main() {
  // allocate memory
  float population_h[POP_SIZE][PARAMS];
  float objectives_h[POP_SIZE][OBJS];
  float utopia_vec_h[OBJS];
  float *population_d;
  float *objectives_d;
  float *utopia_vec_d;

  ERR( hipMalloc( (void**)&population_d, POP_SIZE * PARAMS * sizeof(float) ) );
  ERR( hipMalloc( (void**)&objectives_d, POP_SIZE * OBJS * sizeof(float) ) );
  ERR( hipMalloc( (void**)&utopia_vec_d, OBJS * sizeof(float) ) );

  // create random population
  srand( time( NULL ) );
  for (size_t i = 0; i < POP_SIZE; i++) {
    for (size_t j = 0; j < PARAMS; j++) {
      population_h[i][j] = randomFloat();
    }
  }

  // copy data to GPU
  ERR( hipMemcpy( population_d, population_h, POP_SIZE * PARAMS * sizeof(float), hipMemcpyHostToDevice ) );

  // start the kernel
  mcea<<<1, POP_SIZE>>>( population_d, objectives_d, utopia_vec_d );

  // copy data from GPU
  ERR( hipMemcpy( population_h, population_d, POP_SIZE * PARAMS * sizeof(float), hipMemcpyDeviceToHost ) );
  ERR( hipMemcpy( objectives_h, objectives_d, POP_SIZE * OBJS * sizeof(float), hipMemcpyDeviceToHost ) );
  ERR( hipMemcpy( utopia_vec_h, utopia_vec_d, OBJS * sizeof(float), hipMemcpyDeviceToHost ) );

  // print some solutions
  printVector( population_h[0], PARAMS );
  printVector( objectives_h[0], OBJS );

  // free resources
  ERR( hipFree( population_d ) );
  ERR( hipFree( objectives_d ) );
  ERR( hipFree( utopia_vec_d ) );
}
